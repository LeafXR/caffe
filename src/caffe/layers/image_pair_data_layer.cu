// Copyright 2014 BVLC and contributors.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>
#include <iostream>  // NOLINT(readability/streams)
#include <fstream>  // NOLINT(readability/streams)

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::pair;

namespace caffe {

template <typename Dtype>
Dtype ImagePairDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  CHECK(!pthread_join(thread_, NULL)) << "Pthread joining failed.";
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_data_a_->cpu_data(), sizeof(Dtype) * prefetch_data_a_->count(),
      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
        prefetch_data_b_->cpu_data(), sizeof(Dtype) * prefetch_data_b_->count(),
        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy((*top)[2]->mutable_gpu_data(),
      prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
      hipMemcpyHostToDevice));
  // Start a new prefetch thread
  CHECK(!pthread_create(&thread_, NULL, ImagePairDataLayerPrefetch<Dtype>,
      reinterpret_cast<void*>(this))) << "Pthread execution failed.";
  return Dtype(0.);
}

INSTANTIATE_CLASS(ImagePairDataLayer);

}  // namespace caffe
